#include "hip/hip_runtime.h"
#include <torch/torch.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "hip/hip_vector_types.h"
#include "backwardKernel.h"

#define BLOCK_X 32
#define BLOCK_Y 32
#define BLOCK_A 1
#define PI 3.14159265359
#define CHECK_CUDA(x) AT_ASSERTM(x.type().is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) AT_ASSERTM(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)

texture<float, hipTextureType3D, hipReadModeElementType> sinoTexture;

__device__ float CalcuWeight(float3 pixel, float alpha, float gammamax1, float gammamax2, float Rf,float Rm)
{
	float w, w_ps, w_t;

	float x = pixel.x;
	float y = pixel.y;
    float z = pixel.z;
	float r = sqrt(x * x + y * y);
	float phi = atan2(-1 * x, y);
	float c1 = 1 / tan(gammamax1);
	float c2 = 1 / tan(gammamax2);
	float dr = Rm * Rm / (2 * Rf);  // alpha, Rm, Rf未定
	float theta = alpha - atan2(r * sin(alpha - phi), Rf + r * cos(alpha - phi));
	while (theta < phi - PI)
	{
		theta += 2 * PI;
	}
	while (theta > phi + PI)
	{
		theta -= 2 * PI;
	}
	float theta1 = phi - PI / 2;
	float theta2 = phi + PI / 2;
	float dtheta, angle1, angle2, angle3, angle4;

	//计算体素加权
	w_t = 0;
	w_ps = 0;

    if (Rf - z * c1 < 0)
	{
	    w_t = 1;
		if (r >= z * c1 - Rf)
		{
			dtheta = asin((Rf - z * c1) / r) - atan(sqrt(r * r - (Rf - z * c1)*(Rf - z * c1)) / (z * c1));
			while (dtheta < -PI)
			{
				dtheta += 2 * PI;
			}
			while (dtheta > PI)
			{
				dtheta -= 2 * PI;
			}
			if (dtheta < 0)
			{
				if (abs(theta - phi) <= PI / 2 + dtheta)
				{
					w_ps = 2;// * PI / (PI + 2 * dtheta);
				}
			}
			else
			{
				angle1 = theta1 - dtheta;
				angle2 = theta2 + dtheta;
				angle3 = theta1 + dtheta;
				angle4 = theta2 - dtheta;
				if (theta > angle1 && theta < angle3)
				{
					w_ps = 1 + sin(PI*(theta - angle1 - dtheta) / (2 * dtheta));
				}
				else
				{
					if (theta < angle2 && theta > angle4)
					{
						w_ps = 1 - sin(PI*(theta - angle2 + dtheta) / (2 * dtheta));
					}
					else
					{
						if (theta >= angle3 && theta <= angle4)
						{
							w_ps = 2;
						}
					}
				}
			}
		}
	}
	else
	{
		if (Rf - z * c2 > 0)
		{
			w_t = 1;
			if (r >= Rf - z * c2)
			{
				dtheta = asin((Rf - z * c2) / r) - atan(sqrt(r * r - (Rf - z * c2)*(Rf - z * c2)) / (z * c1));
				while (dtheta < -PI)
				{
					dtheta += 2 * PI;
				}
				while (dtheta > PI)
				{
					dtheta -= 2 * PI;
				}
				if (dtheta > 0)
				{
					if (abs(theta - phi) >= PI / 2 + dtheta)
					{
						w_ps = 2;
					}
				}
				else
				{
					angle1 = theta1 + dtheta;
					angle2 = theta2 - dtheta;
					angle3 = theta1 - dtheta;
					angle4 = theta2 + dtheta;
					if (theta > angle1 && theta < angle3)
					{
						w_ps = 1 - sin(PI*(theta - angle1 - dtheta) / (2 * dtheta));
					}
					else
					{
						if (theta < angle2 && theta > angle4)
						{
							w_ps = 1 + sin(PI*(theta - angle2 + dtheta) / (2 * dtheta));
						}
						else
						{
							if (theta <= angle1 || theta >= angle2)
							{
								w_ps = 2;
							}
						}
					}
				}
			}
		}
		else
		{
			float r1 = Rf - z * c1;
			float r2 = z * c2 - Rf;
			float z1 = (Rf - r) / c1;
			float z2 = (Rf + r) / c2;
			float z0 = 2 * Rf / (c1 + c2);
			if (r >= r1 && r <= r2)
			{
				w_t = 1;
				dtheta = asin((Rf - z * c1) / r) - atan(sqrt(r * r - (Rf - z * c1)*(Rf - z * c1)) / (z * c1));
				while (dtheta < -PI)
				{
					dtheta += 2 * PI;
				}
				while (dtheta > PI)
				{
					dtheta -= 2 * PI;
				}
				if (dtheta < 0)
				{
					if (abs(theta - phi) <= PI / 2 + dtheta)
					{
						w_ps = 2;
					}
				}
				else
				{
					angle1 = theta1 - dtheta;
					angle2 = theta2 + dtheta;
					angle3 = theta1 + dtheta;
					angle4 = theta2 - dtheta;
					if (theta > angle1 && theta < angle3)
					{
						w_ps = 1 + sin(PI*(theta - angle1 - dtheta) / (2 * dtheta));
					}
					else
					{
						if (theta < angle2 && theta > angle4)
						{
							w_ps = 1 - sin(PI*(theta - angle2 + dtheta) / (2 * dtheta));
						}
						else
						{
							if (theta >= angle3 && theta <= angle4)
							{
								w_ps = 2;
							}
						}
					}
				}
			}
			else
			{
				if (r <= r1 && r >= r2)
				{
					w_t = 1;
					dtheta = asin((Rf - z * c2) / r) - atan(sqrt(r * r - (Rf - z * c2)*(Rf - z * c2)) / (z * c2));
					while (dtheta < -PI)
					{
						dtheta += 2 * PI;
					}
					while (dtheta > PI)
					{
						dtheta -= 2 * PI;
					}
					if (dtheta > 0)
					{
						if (abs(theta - phi) >= PI / 2 + dtheta)
						{
							w_ps = 2;
						}
					}
					else
					{
						angle1 = theta1 + dtheta;
						angle2 = theta2 - dtheta;
						angle3 = theta1 - dtheta;
						angle4 = theta2 + dtheta;
						if (theta > angle1 && theta < angle3)
						{
							w_ps = 1 - sin(PI*(theta - angle1 - dtheta) / (2 * dtheta));
						}
						else
						{
							if (theta < angle2 && theta > angle4)
							{
								w_ps = 1 + sin(PI*(theta - angle2 + dtheta) / (2 * dtheta));
							}
							else
							{
								if (theta <= angle1 || theta >= angle2)
								{
									w_ps = 2;
								}
							}
						}
					}
				}
				else
				{
					if (r <= r1 && r <= r2)
					{
						if (r2 >= r1)
						{
							if (r >= r1 - 2 * dr)
							{
								w_t = 0.5 + 0.5 * sin(PI * (r - r1 + dr) / (2 * dr));
								w_ps = 1 + (z/(z1-z0)-z0/(z1-z0)) * cos(theta - phi);
							}
						}
						else
						{
							if (r >= r2 - 2 * dr)
							{
								w_t = 0.5 + 0.5 * sin(PI * (r - r2 + dr) / (2 * dr));
								w_ps = 1 - (z/(z2-z0)-z0/(z2-z0)) * cos(theta - phi);
							}
						}
					}
					else
					{
						w_t = 1;
						float dtheta1 = asin((Rf - z * c1) / r) - atan(sqrt(r * r - (Rf - z * c1)*(Rf - z * c1)) / (z * c1));
						float dtheta2 = asin((Rf - z * c2) / r) - atan(sqrt(r * r - (Rf - z * c2)*(Rf - z * c2)) / (z * c2));
						while (dtheta1 < -PI)
						{
							dtheta1 += 2 * PI;
						}
						while (dtheta1 > PI)
						{
							dtheta1 -= 2 * PI;
						}
						while (dtheta2 < -PI)
						{
							dtheta2 += 2 * PI;
						}
						while (dtheta2 > PI)
						{
							dtheta2 -= 2 * PI;
						}
						if (dtheta1 <= dtheta2)
						{
							w_ps = 0;
						}
						else
						{
							angle1 = theta1 - dtheta1;
							angle2 = theta2 + dtheta1;
							angle3 = theta1 - dtheta2;
							angle4 = theta2 + dtheta2;
							if ((theta > angle1 && theta < angle3) || (theta > angle4 && theta < angle2))
							{
								w_ps = PI / (dtheta1 - dtheta2);
								w_ps = w_ps > 2 ? 2 : w_ps;
							}
						}
					}
				}
			}
		}
	}

	w = 1 - w_t + w_t * w_ps;
    return w;
}

__global__ void backwardKernel(float* volume, const uint3 volumeSize, const uint2 detectorSize, const float* projectVector, const uint index,const int anglesNum,const float3 volumeCenter, const float2 detectorCenter,
                               const float volbiasz, const float dSampleInterval, const float dSliceInterval, const float sourceRadius, const float sourceZpos, const float fBiaz, const float  SID){
    uint3 volumeIdx = make_uint3(blockIdx.x*blockDim.x+threadIdx.x, blockIdx.y*blockDim.y+threadIdx.y, blockIdx.z*blockDim.z+threadIdx.z);
    if (volumeIdx.x >= volumeSize.x || volumeIdx.y >= volumeSize.y){
        return;
    }

    float gammamax1 = atan(abs(fBiaz) / abs(SID + detectorCenter.x));
    float gammamax2 = atan(abs(fBiaz) / abs(SID + detectorCenter.x + detectorSize.x));

    for(int k=0;k<volumeSize.z;k++){
        float value = 0.0f;
        for(int angleIdx = index;angleIdx < index + BLOCK_A;angleIdx++){
            float3 sourcePosition = make_float3(projectVector[angleIdx*12], projectVector[angleIdx*12+1], projectVector[angleIdx*12+2]);
            float3 detectorPosition = make_float3(projectVector[angleIdx*12+3], projectVector[angleIdx*12+4], projectVector[angleIdx*12+5]);
            float3 u = make_float3(projectVector[angleIdx*12+6], projectVector[angleIdx*12+7], projectVector[angleIdx*12+8]);
            float3 v = make_float3(projectVector[angleIdx*12+9], projectVector[angleIdx*12+10], projectVector[angleIdx*12+11]);
            float3 coordinates = make_float3((volumeCenter.x + volumeIdx.x) * dSampleInterval, (volumeCenter.y + volumeIdx.y) * dSampleInterval,(volumeCenter.z + k) * dSliceInterval + volbiasz);
            float fScale = __fdividef(1.0f, det3(u, v, sourcePosition-coordinates));
            fScale = det3(u, v, sourcePosition-coordinates) == 0 ? 0 : fScale;
            float detectorX = fScale * det3(coordinates-sourcePosition,v,sourcePosition-detectorPosition)-detectorCenter.x;
            float detectorY = fScale * det3(u, coordinates-sourcePosition,sourcePosition-detectorPosition)-detectorCenter.y;
            float fr = fScale * det3(u, v, sourcePosition);

            float alpha = angleIdx * 2 * PI / anglesNum - PI / 2;
            float3 pixel = make_float3((volumeCenter.x + volumeIdx.x) * dSampleInterval, (volumeCenter.y + volumeIdx.y) * dSampleInterval, abs((volumeCenter.z + k) * dSliceInterval + volbiasz - sourceZpos));
            //float weight = CalcuWeight(pixel, alpha, gammamax1, gammamax2, sourceRadius, 0.5 * volumeSize.x * dSampleInterval);
            //value += tex3D(sinoTexture, detectorX, detectorY, angleIdx+0.5f);
			value += fr * fr * tex3D(sinoTexture, detectorX, detectorY, angleIdx + 0.5f);
        }
        int idx = k * volumeSize.x * volumeSize.y + volumeIdx.y * volumeSize.x + volumeIdx.x;
		atomicAdd(&volume[idx], value / anglesNum );
		//volume[idx] += value * 2 * PI / anglesNum;
    }
}

torch::Tensor backward(torch::Tensor sino, torch::Tensor _volumeSize, torch::Tensor _detectorSize, torch::Tensor projectVector,
                        const float volbiasz, const float dSampleInterval, const float dSliceInterval,
                        const float sourceRadius, const float sourceZpos, const float fBiaz, const float  SID,
                        const long device){
    CHECK_INPUT(sino);
    CHECK_INPUT(_volumeSize);
    AT_ASSERTM(_volumeSize.size(0) == 3, "volume size's length must be 3");
    CHECK_INPUT(_detectorSize);
    AT_ASSERTM(_detectorSize.size(0) == 2, "detector size's length must be 2");
    CHECK_INPUT(projectVector);
    AT_ASSERTM(projectVector.size(1) == 12, "project vector's shape must be [angle's number, 12]");

    int angles = projectVector.size(0);
    auto out = torch::zeros({sino.size(0), 1, _volumeSize[2].item<int>(), _volumeSize[1].item<int>(), _volumeSize[0].item<int>()}).to(sino.device());
    float* outPtr = out.data<float>();
    float* sinoPtr = sino.data<float>();

    hipSetDevice(device);
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
    sinoTexture.addressMode[0] = hipAddressModeBorder;
    sinoTexture.addressMode[1] = hipAddressModeBorder;
    sinoTexture.addressMode[2] = hipAddressModeBorder;
    sinoTexture.filterMode = hipFilterModeLinear;
    sinoTexture.normalized = false;

    uint3 volumeSize = make_uint3(_volumeSize[0].item<int>(), _volumeSize[1].item<int>(), _volumeSize[2].item<int>());
    uint2 detectorSize = make_uint2(_detectorSize[0].item<int>(), _detectorSize[1].item<int>());
    float3 volumeCenter = make_float3(volumeSize) / -2.0;
    float2 detectorCenter = make_float2(detectorSize) / -2.0;
    for(int batch = 0;batch < sino.size(0); batch++){
        float* sinoPtrPitch = sinoPtr + detectorSize.x * detectorSize.y * angles * batch;
        float* outPtrPitch = outPtr + volumeSize.x * volumeSize.y * volumeSize.z * batch;

        hipExtent m_extent = make_hipExtent(detectorSize.x, detectorSize.y, angles);
        hipArray *sinoArray;
        hipMalloc3DArray(&sinoArray, &channelDesc, m_extent);
        hipMemcpy3DParms copyParams = {0};
        copyParams.srcPtr = make_hipPitchedPtr((void*)sinoPtrPitch, detectorSize.x*sizeof(float), detectorSize.x, detectorSize.y);
        copyParams.dstArray = sinoArray;
        copyParams.kind = hipMemcpyDeviceToDevice;
        copyParams.extent = m_extent;
        hipMemcpy3D(&copyParams);
        hipBindTextureToArray(sinoTexture, sinoArray, channelDesc);

        const dim3 blockSize = dim3(BLOCK_X, BLOCK_Y, 1);
        const dim3 gridSize = dim3(volumeSize.x / BLOCK_X + 1, volumeSize.y / BLOCK_Y + 1, 1);
        for (int angle = 0; angle < angles; angle+=BLOCK_A){
           backwardKernel<<<gridSize, blockSize>>>(outPtrPitch, volumeSize, detectorSize, (float*)projectVector.data<float>(), angle,angles,volumeCenter,detectorCenter,
                                                   volbiasz, dSampleInterval, dSliceInterval, sourceRadius, sourceZpos, fBiaz, SID);
        }
      hipUnbindTexture(sinoTexture);
      hipFreeArray(sinoArray);
    }
    return out;
}

__global__ void backwardKernel_F(float* volume, const uint3 volumeSize, const uint2 detectorSize, const float* projectVector, const uint index, const int anglesNum, const float3 volumeCenter, const float2 detectorCenter,
	const float volbiasz, const float dSampleInterval, const float dSliceInterval, const float sourceRadius, const float sourceZpos, const float fBiaz, const float  SID, const uint systemNum) {
	uint3 volumeIdx = make_uint3(blockIdx.x * blockDim.x + threadIdx.x, blockIdx.y * blockDim.y + threadIdx.y, blockIdx.z * blockDim.z + threadIdx.z);
	if (volumeIdx.x >= volumeSize.x || volumeIdx.y >= volumeSize.y || volumeIdx.z >= systemNum) {
		return;
	}

	float gammamax1 = atan(abs(fBiaz) / abs(SID + detectorCenter.x));
	float gammamax2 = atan(abs(fBiaz) / abs(SID + detectorCenter.x + detectorSize.x));
	float alpha = index * 2 * PI / anglesNum - PI / 2;

	unsigned projectVectorIdx = volumeIdx.z * 12;
	float3 sourcePosition = make_float3(projectVector[projectVectorIdx], projectVector[projectVectorIdx + 1], projectVector[projectVectorIdx + 2]);
	float3 detectorPosition = make_float3(projectVector[projectVectorIdx + 3], projectVector[projectVectorIdx + 4], projectVector[projectVectorIdx + 5]);
	float3 u = make_float3(projectVector[projectVectorIdx + 6], projectVector[projectVectorIdx + 7], projectVector[projectVectorIdx + 8]);
	float3 v = make_float3(projectVector[projectVectorIdx + 9], projectVector[projectVectorIdx + 10], projectVector[projectVectorIdx + 11]);

	for (int k = 0; k < volumeSize.z; k++) {
		float3 coordinates = make_float3((volumeCenter.x + volumeIdx.x) * dSampleInterval, (volumeCenter.y + volumeIdx.y) * dSampleInterval, (volumeCenter.z + k) * dSliceInterval + volbiasz);
		float fScale = __fdividef(1.0f, det3(u, v, sourcePosition - coordinates));
		fScale = det3(u, v, sourcePosition - coordinates) == 0 ? 0 : fScale;
		float detectorX = fScale * det3(coordinates - sourcePosition, v, sourcePosition - detectorPosition) - detectorCenter.x;
		float detectorY = fScale * det3(u, coordinates - sourcePosition, sourcePosition - detectorPosition) - detectorCenter.y;
		float fr = fScale * det3(u, v, sourcePosition);
		float3 pixel = make_float3((volumeCenter.x + volumeIdx.x) * dSampleInterval, (volumeCenter.y + volumeIdx.y) * dSampleInterval, abs((volumeCenter.z + k) * dSliceInterval + volbiasz - sourceZpos));
		//float weight = CalcuWeight(pixel, alpha, gammamax1, gammamax2, sourceRadius, 0.5 * volumeSize.x * dSampleInterval);
		float value = tex3D(sinoTexture, detectorX + 0.5f, detectorY + 0.5f, index + 0.5f);
		//float value =  tex3D(sinoTexture, detectorX + 0.5f, detectorY + 0.5f, index + 0.5f);
		int idx = k * volumeSize.x * volumeSize.y + volumeIdx.y * volumeSize.x + volumeIdx.x;
		atomicAdd(&volume[idx], value / anglesNum / systemNum);
	}
}

void backward_F(torch::Tensor out, torch::Tensor sino, torch::Tensor _volumeSize, torch::Tensor _detectorSize, torch::Tensor projectVector,
	const float volbiasz, const float dSampleInterval, const float dSliceInterval,
	const float sourceRadius, const float sourceZpos, const float fBiaz, const float SID,
	const int systemNum, const long device) {
	CHECK_INPUT(sino);
	CHECK_INPUT(_volumeSize);
	AT_ASSERTM(_volumeSize.size(0) == 3, "volume size's length must be 3");
	CHECK_INPUT(_detectorSize);
	AT_ASSERTM(_detectorSize.size(0) == 2, "detector size's length must be 2");
	CHECK_INPUT(projectVector);
	AT_ASSERTM(projectVector.size(1) == 12, "project vector's shape must be [angle's number, 12]");
	int angles = projectVector.size(0) / systemNum;
	//auto out = torch::zeros({sino.size(0), 1, _volumeSize[2].item<int>(), _volumeSize[1].item<int>(), _volumeSize[0].item<int>()}).to(sino.device());
	float* outPtr = out.data<float>();
	float* sinoPtr = sino.data<float>();

	hipSetDevice(device);
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
	sinoTexture.addressMode[0] = hipAddressModeBorder;
	sinoTexture.addressMode[1] = hipAddressModeBorder;
	sinoTexture.addressMode[2] = hipAddressModeBorder;
	sinoTexture.filterMode = hipFilterModeLinear;
	sinoTexture.normalized = false;

	uint3 volumeSize = make_uint3(_volumeSize[0].item<int>(), _volumeSize[1].item<int>(), _volumeSize[2].item<int>());
	uint2 detectorSize = make_uint2(_detectorSize[0].item<int>(), _detectorSize[1].item<int>());
	float3 volumeCenter = make_float3(volumeSize) / -2.0;
	float2 detectorCenter = make_float2(detectorSize) / -2.0;

	for (int batch = 0; batch < sino.size(0); batch++) {
		float* sinoPtrPitch = sinoPtr + detectorSize.x * detectorSize.y * angles * batch;
		float* outPtrPitch = outPtr + volumeSize.x * volumeSize.y * volumeSize.z * batch;

		hipExtent m_extent = make_hipExtent(detectorSize.x, detectorSize.y, angles);
		hipArray* sinoArray;
		hipMalloc3DArray(&sinoArray, &channelDesc, m_extent);
		hipMemcpy3DParms copyParams = { 0 };
		copyParams.srcPtr = make_hipPitchedPtr((void*)sinoPtrPitch, detectorSize.x * sizeof(float), detectorSize.x, detectorSize.y);
		copyParams.dstArray = sinoArray;
		copyParams.kind = hipMemcpyDeviceToDevice;
		copyParams.extent = m_extent;
		hipMemcpy3D(&copyParams);
		hipBindTextureToArray(sinoTexture, sinoArray, channelDesc);

		const dim3 blockSize = dim3(BLOCK_X, BLOCK_Y, BLOCK_A);
		const dim3 gridSize = dim3(volumeSize.x / BLOCK_X + 1, volumeSize.y / BLOCK_Y + 1, systemNum / BLOCK_A );
		auto projVec = projectVector.reshape({ angles, systemNum * 12 });
		for (int angle = 0; angle < angles; angle++) {
			backwardKernel_F << <gridSize, blockSize >> > (outPtrPitch, volumeSize, detectorSize, (float*)projVec[angle].data<float>(), angle, angles, volumeCenter, detectorCenter,
				volbiasz, dSampleInterval, dSliceInterval, sourceRadius, sourceZpos, fBiaz, SID, systemNum);
			hipDeviceSynchronize();
		}
		hipUnbindTexture(sinoTexture);
		hipFreeArray(sinoArray);
	}
	//return out;
}

__global__ void rotationKernal(float* out, const uint2 detectorSize, const uint2 targetdetectorSize, const float* projectVector,
	const float* targetprojectVector, const uint index, const int anglesNum, const float2 detectorCenter, const float2 targetdetectorCenter)
{ 
	uint3 targetdetectorIdx = make_uint3(blockIdx.x * blockDim.x + threadIdx.x, blockIdx.y * blockDim.y + threadIdx.y, blockIdx.z * blockDim.z + threadIdx.z);
	if (targetdetectorIdx.x >= targetdetectorSize.x || targetdetectorIdx.y >= targetdetectorSize.y ) {
		return;
	}
	float3 sourcePosition = make_float3(projectVector[0], projectVector[1], projectVector[2]);
	float3 detectorPosition = make_float3(projectVector[3], projectVector[4], projectVector[5]);
	float3 u = make_float3(projectVector[6], projectVector[7], projectVector[8]);
	float3 v = make_float3(projectVector[9], projectVector[10], projectVector[11]);
	float3 targetdetectorPosition = make_float3(targetprojectVector[3], targetprojectVector[4], targetprojectVector[5]);
	float3 targetu = make_float3(targetprojectVector[6], targetprojectVector[7], targetprojectVector[8]);
	float3 targetv = make_float3(targetprojectVector[9], targetprojectVector[10], targetprojectVector[11]);
	float targetdetectorX = targetdetectorIdx.x + targetdetectorCenter.x;
	float targetdetectorY = targetdetectorIdx.y + targetdetectorCenter.y;
	float3 coordinates = targetdetectorPosition+ targetdetectorX* targetu+ targetdetectorY* targetv;
	float fScale = 0;
	if ((det3(u, v, sourcePosition - coordinates)) != 0) { fScale = __fdividef(1.0f, det3(u, v, sourcePosition - coordinates)); }
	float detectorX = fScale * det3(coordinates - sourcePosition, v, sourcePosition - detectorPosition) - detectorCenter.x;
	float detectorY = fScale * det3(u, coordinates - sourcePosition, sourcePosition - detectorPosition) - detectorCenter.y;
	float value = tex3D(sinoTexture, detectorX + 0.5f, detectorY + 0.5f, index + 0.5f);
	unsigned sinogramIdx = index * targetdetectorSize.x * targetdetectorSize.y + targetdetectorIdx.y * targetdetectorSize.x + targetdetectorIdx.x;
	atomicAdd(&out[sinogramIdx], value);
}


void rotation(torch::Tensor out, torch::Tensor sino, torch::Tensor _detectorSize, torch::Tensor _detectorSize1, torch::Tensor projectVector,
	torch::Tensor projectVector1, const long device)
{
	CHECK_INPUT(sino);
	CHECK_INPUT(_detectorSize);
	AT_ASSERTM(_detectorSize.size(0) == 2, "detector size's length must be 3");
	CHECK_INPUT(_detectorSize1);
	AT_ASSERTM(_detectorSize.size(0) == 2, "detector size's length must be 2");
	CHECK_INPUT(projectVector);
	AT_ASSERTM(projectVector.size(1) == 12, "project vector's shape must be [angle's number, 12]");
	int angles = projectVector.size(0);
	out.zero_();
	float* outPtr = out.data<float>();
	float* sinoPtr = sino.data<float>();

	hipSetDevice(device);
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
	sinoTexture.addressMode[0] = hipAddressModeBorder;
	sinoTexture.addressMode[1] = hipAddressModeBorder;
	sinoTexture.addressMode[2] = hipAddressModeBorder;
	sinoTexture.filterMode = hipFilterModeLinear;
	sinoTexture.normalized = false;

	uint2 detectorSize = make_uint2(_detectorSize[0].item<int>(), _detectorSize[1].item<int>());
	float2 detectorCenter = make_float2(detectorSize) / -2.0;
	uint2 detectorSize1 = make_uint2(_detectorSize1[0].item<int>(), _detectorSize1[1].item<int>());
	float2 detectorCenter1 = make_float2(detectorSize1) / -2.0;

	for (int batch = 0; batch < sino.size(0); batch++) {
		float* sinoPtrPitch = sinoPtr + detectorSize.x * detectorSize.y * angles * batch;
		float* outPtrPitch = outPtr + detectorSize1.x * detectorSize1.y * angles * batch;

		hipExtent m_extent = make_hipExtent(detectorSize.x, detectorSize.y, angles);
		hipArray* sinoArray;
		hipMalloc3DArray(&sinoArray, &channelDesc, m_extent);
		hipMemcpy3DParms copyParams = { 0 };
		copyParams.srcPtr = make_hipPitchedPtr((void*)sinoPtrPitch, detectorSize.x * sizeof(float), detectorSize.x, detectorSize.y);
		copyParams.dstArray = sinoArray;
		copyParams.kind = hipMemcpyDeviceToDevice;
		copyParams.extent = m_extent;
		hipMemcpy3D(&copyParams);
		hipBindTextureToArray(sinoTexture, sinoArray, channelDesc);

		const dim3 blockSize = dim3(BLOCK_X, BLOCK_Y, BLOCK_A);
		const dim3 gridSize = dim3(detectorSize1.x / BLOCK_X + 1, detectorSize1.y / BLOCK_Y + 1,  BLOCK_A);
		auto projVec = projectVector.reshape({ angles, 12 });
		auto targetprojVec = projectVector1.reshape({ angles, 12 });
		for (int angle = 0; angle < angles; angle++) {
			rotationKernal << <gridSize, blockSize >> > (outPtrPitch, detectorSize, detectorSize1, (float*)projVec[angle].data<float>(), (float*)targetprojVec[angle].data<float>(),
				angle, angles, detectorCenter, detectorCenter1);
			hipDeviceSynchronize();
		}
		hipUnbindTexture(sinoTexture);
		hipFreeArray(sinoArray);
	}
}

__global__ void backwardKernel_P(float* volume, const uint3 volumeSize, const uint2 detectorSize, const float* projectMatrix, const uint index, const int anglesNum, const float3 volumeCenter, const float2 detectorCenter,
	const float volbiasz, const float dSampleInterval, const float dSliceInterval) {
	uint3 volumeIdx = make_uint3(blockIdx.x * blockDim.x + threadIdx.x, blockIdx.y * blockDim.y + threadIdx.y, blockIdx.z * blockDim.z + threadIdx.z);
	if (volumeIdx.x >= volumeSize.x || volumeIdx.y >= volumeSize.y) {
		return;
	}
	for (int k = 0; k < volumeSize.z; k++) {
		float value = 0.0f;
		for (int angleIdx = index; angleIdx < index + BLOCK_A; angleIdx++) {
			// coordinates * projectMatrix to get the detector piexl
			// 
			float4 Hrow1 = make_float4(projectMatrix[angleIdx * 12], projectMatrix[angleIdx * 12 + 3], projectMatrix[angleIdx * 12 + 6], projectMatrix[angleIdx * 12 + 9]);
			float4 Hrow2 = make_float4(projectMatrix[angleIdx * 12 + 1], projectMatrix[angleIdx * 12 + 4], projectMatrix[angleIdx * 12 + 7], projectMatrix[angleIdx * 12 + 10]);
			float4 Hrow3 = make_float4(projectMatrix[angleIdx * 12 + 2], projectMatrix[angleIdx * 12 + 5], projectMatrix[angleIdx * 12 + 8], projectMatrix[angleIdx * 12 + 11]);
			float4 coordinates = make_float4((volumeCenter.x + volumeIdx.x) * dSampleInterval, (volumeCenter.y + volumeIdx.y) * dSampleInterval, (volumeCenter.z + k) * dSliceInterval + volbiasz, 1.0);
			float3 pos2D = make_float3(dot(Hrow1, coordinates), dot(Hrow2, coordinates), dot(Hrow3, coordinates));
			float detectorX = 0, detectorY = 0;
			if (pos2D.z != 0) {
				detectorX = pos2D.x / pos2D.z - detectorCenter.x;
				detectorY = pos2D.y / pos2D.z - detectorCenter.y;
			}
			value += tex3D(sinoTexture, detectorX + 0.5f, detectorY + 0.5f, angleIdx + 0.5f);
		}
		int idx = k * volumeSize.x * volumeSize.y + volumeIdx.y * volumeSize.x + volumeIdx.x;
		atomicAdd(&volume[idx], value / anglesNum);
		//volume[idx] += value * 2 * PI / anglesNum;
	}
}

void backward_P(torch::Tensor out,torch::Tensor sino, torch::Tensor _volumeSize, torch::Tensor _detectorSize, torch::Tensor projectMatrix,
	const float volbiasz, const float dSampleInterval, const float dSliceInterval, const long device) {
	CHECK_INPUT(sino);
	CHECK_INPUT(_volumeSize);
	AT_ASSERTM(_volumeSize.size(0) == 3, "volume size's length must be 3");
	CHECK_INPUT(_detectorSize);
	AT_ASSERTM(_detectorSize.size(0) == 2, "detector size's length must be 2");
	CHECK_INPUT(projectMatrix);
	AT_ASSERTM(projectMatrix.size(1) == 12, "project vector's shape must be [angle's number, 12]");

	int angles = projectMatrix.size(0);
	out.zero_();
	float* outPtr = out.data<float>();
	float* sinoPtr = sino.data<float>();

	hipSetDevice(device);
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
	sinoTexture.addressMode[0] = hipAddressModeBorder;
	sinoTexture.addressMode[1] = hipAddressModeBorder;
	sinoTexture.addressMode[2] = hipAddressModeBorder;
	sinoTexture.filterMode = hipFilterModeLinear;
	sinoTexture.normalized = false;

	uint3 volumeSize = make_uint3(_volumeSize[0].item<int>(), _volumeSize[1].item<int>(), _volumeSize[2].item<int>());
	uint2 detectorSize = make_uint2(_detectorSize[0].item<int>(), _detectorSize[1].item<int>());
	float3 volumeCenter = make_float3(volumeSize) / -2.0;
	float2 detectorCenter = make_float2(detectorSize) / -2.0;
	for (int batch = 0; batch < sino.size(0); batch++) {
		float* sinoPtrPitch = sinoPtr + detectorSize.x * detectorSize.y * angles * batch;
		float* outPtrPitch = outPtr + volumeSize.x * volumeSize.y * volumeSize.z * batch;

		hipExtent m_extent = make_hipExtent(detectorSize.x, detectorSize.y, angles);
		hipArray* sinoArray;
		hipMalloc3DArray(&sinoArray, &channelDesc, m_extent);
		hipMemcpy3DParms copyParams = { 0 };
		copyParams.srcPtr = make_hipPitchedPtr((void*)sinoPtrPitch, detectorSize.x * sizeof(float), detectorSize.x, detectorSize.y);
		copyParams.dstArray = sinoArray;
		copyParams.kind = hipMemcpyDeviceToDevice;
		copyParams.extent = m_extent;
		hipMemcpy3D(&copyParams);
		hipBindTextureToArray(sinoTexture, sinoArray, channelDesc);

		const dim3 blockSize = dim3(BLOCK_X, BLOCK_Y, 1);
		const dim3 gridSize = dim3(volumeSize.x / BLOCK_X + 1, volumeSize.y / BLOCK_Y + 1, 1);
		for (int angle = 0; angle < angles; angle += BLOCK_A) {
			backwardKernel_P << <gridSize, blockSize >> > (outPtrPitch, volumeSize, detectorSize, (float*)projectMatrix.data<float>(), angle, angles, volumeCenter, detectorCenter,
				volbiasz, dSampleInterval, dSliceInterval);
		}
		hipUnbindTexture(sinoTexture);
		hipFreeArray(sinoArray);
	}
}
